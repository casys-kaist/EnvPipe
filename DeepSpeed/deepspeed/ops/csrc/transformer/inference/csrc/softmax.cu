#include "hip/hip_runtime.h"
/*
Copyright 2022 The Microsoft DeepSpeed Team
*/

#include <limits>
#include "inference_cuda_layers.h"

#ifndef __HIP_PLATFORM_HCC__
#include <hip/hip_runtime_api.h>
#endif
#include <cstdio>
#include <cstdlib>
#include <ctime>

#define ATTN_THREADS 256
#define MAX_REG_SIZE 8

#define minus_infinity -10000.0

void CheckCudaErrorAux(const char* file, unsigned line)
{
    hipError_t err = hipGetLastError();
    if (err == hipSuccess) return;
    std::cerr << hipGetErrorString(err) << "(" << err << ") at " << file << ":" << line
              << std::endl;
    throw std::runtime_error("CUDA ERROR!!!\n");
}

#define CUDA_CHECK_ERROR() CheckCudaErrorAux(__FILE__, __LINE__)

namespace cg = cooperative_groups;

__global__ void attn_softmax_v2(__half* vals,
                                __half* mask,
                                __half* alibi,
                                float layer_scale,
                                bool triangular,
                                bool recompute,
                                bool local_attention,
                                int window_size,
                                int total_count,
                                int heads,
                                int sequence_length,
                                int num_seq,
                                int head_offset,
                                int mask_stride,
                                int mp_size,
                                int iterations,
                                int reduceWidth)
{
#ifdef HALF_PRECISION_AVAILABLE

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    float2 low_data[MAX_REG_SIZE];
    float2 high_data[MAX_REG_SIZE];
    const __half zero_h = __float2half(0.f);

    int wid = threadIdx.x >> 5;
    int lane = threadIdx.x & 0x1f;
    int warp_num = blockDim.x >> 5;

    int reduce_blocks = reduceWidth >> 5;
    int seq_lane = threadIdx.x % reduceWidth;

    __shared__ float partialSum[MAX_WARP_NUM];

    int iter_offset = blockIdx.x * (warp_num / reduce_blocks) + (wid / reduce_blocks);
    int batch_idx = iter_offset / (num_seq * heads);
    int alibi_offset = batch_idx * heads * mp_size + head_offset;
    int mask_offset = batch_idx * mask_stride + (iter_offset % mask_stride);

    if (iter_offset < total_count) {
        vals += (iter_offset * sequence_length);

        alibi_offset = (alibi_offset + ((iter_offset / num_seq) % heads)) * sequence_length;
        mask_offset = mask_offset * sequence_length;
        int seq_id = iter_offset % num_seq;
        int seq_id4 = seq_id >> 2;

        int real_seq_id = seq_id + (num_seq == sequence_length ? 0 : sequence_length);
        int window_stride4 = (local_attention && (real_seq_id >> 2) > (window_size >> 2))
                                 ? (real_seq_id >> 2) - (window_size >> 2)
                                 : 0;
        int window_stride =
            (local_attention && real_seq_id >= window_size) ? real_seq_id - window_size : -1;

        float max_val = minus_infinity;
        // if (lane == 0) printf("%d, %d: %d \n", wid, blockIdx.x, mask_offset);
        for (int i = 0; i < iterations; i++) {
            int data_id = i * (reduceWidth << 2) + (seq_lane << 2);
            if ((!triangular || ((data_id >> 2) <= seq_id4)) && (data_id >> 2) >= window_stride4 &&
                data_id < sequence_length) {
                if ((sequence_length - data_id) >= 4) {
                    low_data[i].x = data_id > window_stride
                                        ? __half2float(vals[data_id]) * layer_scale
                                        : minus_infinity;
                    low_data[i].y = ((!triangular || ((data_id + 1) <= seq_id)) &&
                                     (data_id + 1) > window_stride)
                                        ? __half2float(vals[data_id + 1]) * layer_scale
                                        : minus_infinity;
                    high_data[i].x = ((!triangular || ((data_id + 2) <= seq_id)) &&
                                      (data_id + 2) > window_stride)
                                         ? __half2float(vals[data_id + 2]) * layer_scale
                                         : minus_infinity;
                    high_data[i].y = ((!triangular || ((data_id + 3) <= seq_id)) &&
                                      (data_id + 3) > window_stride)
                                         ? __half2float(vals[data_id + 3]) * layer_scale
                                         : minus_infinity;
                    if (alibi) {
                        low_data[i].x = low_data[i].x + __half2float(alibi[data_id + alibi_offset]);
                        low_data[i].y =
                            low_data[i].y + __half2float(alibi[data_id + alibi_offset + 1]);
                        high_data[i].x =
                            high_data[i].x + __half2float(alibi[data_id + alibi_offset + 2]);
                        high_data[i].y =
                            high_data[i].y + __half2float(alibi[data_id + alibi_offset + 3]);
                    }
                    if (mask) {
                        low_data[i].x += __half2float(mask[data_id + mask_offset]);
                        low_data[i].y += __half2float(mask[data_id + mask_offset + 1]);
                        high_data[i].x += __half2float(mask[data_id + mask_offset + 2]);
                        high_data[i].y += __half2float(mask[data_id + mask_offset + 3]);
                    }
                } else {
                    low_data[i].x = data_id > window_stride
                                        ? __half2float(vals[data_id]) * layer_scale
                                        : minus_infinity;
                    low_data[i].y = (((!triangular || (data_id + 1) <= seq_id) &&
                                      (data_id + 1) > window_stride) &&
                                     (data_id + 1) < sequence_length)
                                        ? __half2float(vals[data_id + 1]) * layer_scale
                                        : minus_infinity;
                    high_data[i].x = (((!triangular || (data_id + 2) <= seq_id) &&
                                       (data_id + 2) > window_stride) &&
                                      (data_id + 2) < sequence_length)
                                         ? __half2float(vals[data_id + 2]) * layer_scale
                                         : minus_infinity;
                    if (alibi) {
                        low_data[i].x = low_data[i].x + __half2float(alibi[data_id + alibi_offset]);
                        if ((data_id + 1) < sequence_length)
                            low_data[i].y =
                                low_data[i].y + __half2float(alibi[data_id + alibi_offset + 1]);
                        if ((data_id + 2) < sequence_length)
                            high_data[i].x =
                                high_data[i].x + __half2float(alibi[data_id + alibi_offset + 2]);
                    }
                    high_data[i].y = minus_infinity;
                    if (mask) {
                        low_data[i].x += __half2float(mask[data_id + mask_offset]);
                        if ((data_id + 1) < sequence_length)
                            low_data[i].y += __half2float(mask[data_id + mask_offset + 1]);
                        if ((data_id + 2) < sequence_length)
                            high_data[i].x += __half2float(mask[data_id + mask_offset + 2]);
                    }
                }
                // if(lane == 0) printf("%f , %d, %d \n", low_data[i].x, data_id, seq_id);
                max_val = (low_data[i].x > max_val ? low_data[i].x : max_val);
                max_val = (low_data[i].y > max_val ? low_data[i].y : max_val);
                max_val = (high_data[i].x > max_val ? high_data[i].x : max_val);
                max_val = (high_data[i].y > max_val ? high_data[i].y : max_val);
            } else {
                low_data[i].x = minus_infinity;
                low_data[i].y = minus_infinity;
                high_data[i].x = minus_infinity;
                high_data[i].y = minus_infinity;
            }
        }

        for (int i = 1; i < WARP_SIZE; i *= 2) {
            auto temp = g.shfl_xor(max_val, i);
            max_val = (temp > max_val ? temp : max_val);
        }

        if (reduceWidth > WARP_SIZE) {
            if (lane == 0) partialSum[wid] = max_val;
            b.sync();

            if (lane < warp_num) max_val = partialSum[lane];

            b.sync();

            for (int i = 1; i < reduce_blocks; i *= 2) {
                auto temp = g.shfl_xor(max_val, i);
                max_val = (temp > max_val ? temp : max_val);
            }

            max_val = g.shfl(max_val, threadIdx.x / WARP_SIZE);
        }
        float sum = 0;
        for (int i = 0; i < iterations; i++) {
            low_data[i].x = __expf(low_data[i].x - max_val);
            low_data[i].y = __expf(low_data[i].y - max_val);
            high_data[i].x = __expf(high_data[i].x - max_val);
            high_data[i].y = __expf(high_data[i].y - max_val);

            sum += (low_data[i].x + low_data[i].y + high_data[i].x + high_data[i].y);
        }

        for (int i = 1; i < WARP_SIZE; i *= 2) sum += g.shfl_xor(sum, i);

        if (reduceWidth > WARP_SIZE) {
            if (lane == 0) partialSum[wid] = sum;
            b.sync();

            if (lane < warp_num) sum = partialSum[lane];

            b.sync();

            for (int i = 1; i < reduce_blocks; i *= 2) { sum += g.shfl_xor(sum, i); }

            sum = g.shfl(sum, threadIdx.x / WARP_SIZE);
        }
        sum += 1e-6;
        for (int i = 0; i < iterations; i++) {
            int data_id = i * (reduceWidth << 2) + (seq_lane << 2);

            if (data_id < sequence_length) {
                if ((sequence_length - data_id) >= 4) {
                    vals[data_id] = __float2half(low_data[i].x / sum);
                    vals[data_id + 1] = __float2half(low_data[i].y / sum);
                    vals[data_id + 2] = __float2half(high_data[i].x / sum);
                    vals[data_id + 3] = __float2half(high_data[i].y / sum);
                } else {
                    vals[data_id] = __float2half(low_data[i].x / sum);
                    if ((data_id + 1) < sequence_length)
                        vals[data_id + 1] = __float2half(low_data[i].y / sum);
                    if ((data_id + 2) < sequence_length)
                        vals[data_id + 2] = __float2half(high_data[i].x / sum);
                }
            }
        }
    }
#endif
}

__global__ void attn_softmax_v2(float* vals,
                                float* attn_mask,
                                float* alibi,
                                float layer_scale,
                                bool triangular,
                                bool recompute,
                                bool local_attention,
                                int window_size,
                                int total_count,
                                int heads,
                                int sequence_length,
                                int num_seq,
                                int head_offset,
                                int mask_stride,
                                int mp_size,
                                int iterations,
                                int reduceWidth)
{
    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    float4 data[MAX_REG_SIZE];

    int wid = threadIdx.x >> 5;
    int lane = threadIdx.x & 0x1f;
    int warp_num = blockDim.x >> 5;

    int reduce_blocks = reduceWidth >> 5;
    int seq_lane = threadIdx.x % reduceWidth;

    __shared__ float partialSum[MAX_WARP_NUM];

    int iter_offset = blockIdx.x * (warp_num / reduce_blocks) + (wid / reduce_blocks);
    if (iter_offset < total_count) {
        vals += (iter_offset * sequence_length);

        int batch_idx = iter_offset / (num_seq * heads);
        int alibi_offset = batch_idx * heads * mp_size + head_offset;
        int mask_offset = batch_idx * mask_stride + (iter_offset % mask_stride);
        mask_offset = mask_offset * sequence_length;
        int seq_id = iter_offset % num_seq;
        int seq_id4 = seq_id >> 2;

        int real_seq_id = seq_id + (num_seq == sequence_length ? 0 : sequence_length);
        int window_stride4 = (local_attention && (real_seq_id >> 2) > (window_size >> 2))
                                 ? (real_seq_id >> 2) - (window_size >> 2)
                                 : 0;
        int window_stride =
            (local_attention && real_seq_id >= window_size) ? real_seq_id - window_size : -1;

        float max_val = minus_infinity;

        for (int i = 0; i < iterations; i++) {
            int data_id = i * (reduceWidth << 2) + (seq_lane << 2);
            if ((!triangular || ((data_id >> 2) <= seq_id4)) && (data_id >> 2) >= window_stride4 &&
                data_id < sequence_length) {
                if ((sequence_length - data_id) >= 4) {
                    data[i].x = (data_id > window_stride ? vals[data_id] : minus_infinity);
                    data[i].y = ((!triangular || ((data_id + 1) <= seq_id)) &&
                                 (data_id + 1) > window_stride)
                                    ? vals[data_id + 1]
                                    : minus_infinity;
                    data[i].z = ((!triangular || ((data_id + 2) <= seq_id)) &&
                                 (data_id + 2) > window_stride)
                                    ? vals[data_id + 2]
                                    : minus_infinity;
                    data[i].w = ((!triangular || ((data_id + 3) <= seq_id)) &&
                                 (data_id + 3) > window_stride)
                                    ? vals[data_id + 3]
                                    : minus_infinity;
                    if (attn_mask) {
                        data[i].x += attn_mask[data_id + mask_offset];
                        data[i].y += attn_mask[data_id + mask_offset + 1];
                        data[i].z += attn_mask[data_id + mask_offset + 2];
                        data[i].w += attn_mask[data_id + mask_offset + 3];
                    }
                } else {
                    data[i].x = data_id > window_stride ? vals[data_id] : minus_infinity;
                    data[i].y = (((!triangular || (data_id + 1) <= seq_id)) &&
                                 (data_id + 1) > window_stride && (data_id + 1) < sequence_length)
                                    ? (vals[data_id + 1])
                                    : minus_infinity;
                    data[i].z = (((!triangular || (data_id + 2) <= seq_id)) &&
                                 (data_id + 2) > window_stride && (data_id + 2) < sequence_length)
                                    ? (vals[data_id + 2])
                                    : minus_infinity;
                    data[i].w = minus_infinity;
                    if (attn_mask) {
                        data[i].x += attn_mask[data_id + mask_offset];
                        if ((data_id + 1) < sequence_length)
                            data[i].y += attn_mask[data_id + mask_offset + 1];
                        if ((data_id + 2) < sequence_length)
                            data[i].z += attn_mask[data_id + mask_offset + 2];
                    }
                }
                max_val = (data[i].x > max_val ? data[i].x : max_val);
                max_val = (data[i].y > max_val ? data[i].y : max_val);
                max_val = (data[i].z > max_val ? data[i].z : max_val);
                max_val = (data[i].w > max_val ? data[i].w : max_val);
            } else {
                data[i].x = minus_infinity;
                data[i].y = minus_infinity;
                data[i].z = minus_infinity;
                data[i].w = minus_infinity;
            }
        }

        for (int i = 1; i < WARP_SIZE; i *= 2) {
            auto temp = g.shfl_xor(max_val, i);
            max_val = (temp > max_val ? temp : max_val);
        }

        if (reduceWidth > WARP_SIZE) {
            if (lane == 0) partialSum[wid] = max_val;
            b.sync();

            if (lane < warp_num) max_val = partialSum[lane];

            b.sync();

            for (int i = 1; i < reduce_blocks; i *= 2) {
                auto temp = g.shfl_xor(max_val, i);
                max_val = (temp > max_val ? temp : max_val);
            }

            max_val = g.shfl(max_val, threadIdx.x / WARP_SIZE);
        }

        float sum = 0;
        for (int i = 0; i < iterations; i++) {
            data[i].x = __expf(data[i].x - max_val);
            data[i].y = __expf(data[i].y - max_val);
            data[i].z = __expf(data[i].z - max_val);
            data[i].w = __expf(data[i].w - max_val);

            sum += (data[i].x + data[i].y + data[i].z + data[i].w);
        }

        for (int i = 1; i < WARP_SIZE; i *= 2) sum += g.shfl_xor(sum, i);

        if (reduceWidth > WARP_SIZE) {
            if (lane == 0) partialSum[wid] = sum;
            b.sync();

            if (lane < warp_num) sum = partialSum[lane];

            b.sync();

            for (int i = 1; i < reduce_blocks; i *= 2) { sum += g.shfl_xor(sum, i); }

            sum = g.shfl(sum, threadIdx.x / WARP_SIZE);
        }
        sum += 1e-6;

        for (int i = 0; i < iterations; i++) {
            int data_id = i * (reduceWidth << 2) + (seq_lane << 2);

            if (data_id < sequence_length) {
                if ((sequence_length - data_id) >= 4) {
                    vals[data_id] = data[i].x / sum;
                    vals[data_id + 1] = data[i].y / sum;
                    vals[data_id + 2] = data[i].z / sum;
                    vals[data_id + 3] = data[i].w / sum;
                } else {
                    vals[data_id] = data[i].x / sum;
                    if ((data_id + 1) < sequence_length) vals[data_id + 1] = data[i].y / sum;
                    if ((data_id + 2) < sequence_length) vals[data_id + 2] = data[i].z / sum;
                }
            }
        }
    }
}

template <typename T>
void launch_attn_softmax_v2(T* vals,
                            T* mask,
                            T* alibi,
                            float layer_scale,
                            bool triangular,
                            bool recompute,
                            bool local_attention,
                            int window_size,
                            int batch_size,
                            int heads,
                            int num_seq,
                            int sequence_length,
                            int head_offset,
                            int mask_stride,
                            int mp_size,
                            hipStream_t stream)
{
    int total_count = batch_size * heads * num_seq;
    int warp_num = ATTN_THREADS / WARP_SIZE;
    dim3 grid_dim((total_count - 1) / (warp_num / ((sequence_length - 1) / ATTN_THREADS + 1)) + 1);
    dim3 block_dim(ATTN_THREADS);

    const int reduce_width = ((sequence_length - 1) / ATTN_THREADS + 1) * WARP_SIZE;
    const int iterations = (sequence_length - 1) / (reduce_width << 2) + 1;

    if (sequence_length <= 32768)
        attn_softmax_v2<<<grid_dim, block_dim, 0, stream>>>(vals,
                                                            mask,
                                                            alibi,
                                                            layer_scale,
                                                            triangular,
                                                            recompute,
                                                            local_attention,
                                                            window_size,
                                                            total_count,
                                                            heads,
                                                            sequence_length,
                                                            num_seq,
                                                            head_offset,
                                                            mask_stride,
                                                            mp_size,
                                                            iterations,
                                                            reduce_width);
    else
        throw std::runtime_error("Unsupport Seq_Length!");
}

template void launch_attn_softmax_v2(float* vals,
                                     float* mask,
                                     float* alibi,
                                     float layer_scale,
                                     bool triangular,
                                     bool recompute,
                                     bool local_attention,
                                     int window_size,
                                     int batch_size,
                                     int heads,
                                     int num_seq,
                                     int sequence_length,
                                     int head_offset,
                                     int mask_stride,
                                     int mp_size,
                                     hipStream_t stream);
template void launch_attn_softmax_v2(__half* vals,
                                     __half* mask,
                                     __half* alibi,
                                     float layer_scale,
                                     bool triangular,
                                     bool recompute,
                                     bool local_attention,
                                     int window_size,
                                     int batch_size,
                                     int heads,
                                     int num_seq,
                                     int sequence_length,
                                     int head_offset,
                                     int mask_stride,
                                     int mp_size,
                                     hipStream_t stream);
